#include "hip/hip_runtime.h"
#include "color_space.h"
#include "utility.h"
#include <hip/hip_runtime.h>

using namespace std;

__constant__ float device_YCbCr_matrix[3][3];
__constant__ float device_shift_vector[3];

float* RGB_2_YCbCr(Image& rgb_image){
    int height = rgb_image.height;
    int width = rgb_image.width;
    int channels = rgb_image.channels;
    float* ycbcr_image = new float[height * width * channels];

    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            for (int c = 0; c < channels; c++)
            {
                int index = y * width * channels + x * channels + c;
                int R = rgb_image.data[y][x][0];
                int G = rgb_image.data[y][x][1];
                int B = rgb_image.data[y][x][2];

                ycbcr_image[index] = R * YCbCr_matrix[c][0] +
                                     G * YCbCr_matrix[c][1] +
                                     B * YCbCr_matrix[c][2] +
                                     shift_vector[c];
            }
        }
    }

    return ycbcr_image;
}

float* YcbCr_2_RGB(float* ycbcr_image, int height, int width, int channels){
    float* rgb_image = new float[height * width * channels];

    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            for (int c = 0; c < channels; c++)
            {
                int ycbcr_index = y * width * channels + x * channels;
                int index = y * width * channels + x * channels + c;
                float Y = ycbcr_image[ycbcr_index + 0];
                float Cb = ycbcr_image[ycbcr_index + 1];
                float Cr = ycbcr_image[ycbcr_index + 2];

                rgb_image[index] = (Y - shift_vector[0]) * inv_YCbCr_matrix[c][0] +
                                   (Cb - shift_vector[1]) * inv_YCbCr_matrix[c][1] +
                                   (Cr - shift_vector[2]) * inv_YCbCr_matrix[c][2];
                                   
            }
        }
    }

    return rgb_image;                           
}

float* chrominance_subsample(float* ycbcr_image, int height, int width, int channels){
    int CbCr_height = height / 2;
    int CbCr_width = width / 2;
    float* subsampled_image = new float[height * width + 2 * CbCr_height * CbCr_width];

    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            int origin_index = y * width * channels + x * channels;
            int Y_index = y * width + x;
            subsampled_image[Y_index] = ycbcr_image[origin_index];
            
            // subsampling ratio: 4:2:0, so we only need to sample CbCr once every four pixels
            if(y % 2 == 0 && x % 2 == 0){
                int Cb_index = height * width + (y / 2) * CbCr_width + (x / 2);
                int Cr_index = height * width + CbCr_height * CbCr_width + (y / 2) * CbCr_width + (x / 2);
                subsampled_image[Cb_index] = ycbcr_image[origin_index + 1];
                subsampled_image[Cr_index] = ycbcr_image[origin_index + 2];
            }
        }
    }

    return subsampled_image;
}

float* chrominance_upsample(float* subsampled_image, int height, int width, int channels){
    int CbCr_height = height / 2;
    int CbCr_width = width / 2;
    float* ycbcr_image = new float[height * width * channels];

    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            int origin_index = y * width * channels + x * channels;
            int Y_index = y * width + x;
            int Cb_index = height * width + (y / 2) * CbCr_width + (x / 2);
            int Cr_index = height * width + CbCr_height * CbCr_width + (y / 2) * CbCr_width + (x / 2);
            ycbcr_image[origin_index + 0] = subsampled_image[Y_index];
            ycbcr_image[origin_index + 1] = subsampled_image[Cb_index];
            ycbcr_image[origin_index + 2] = subsampled_image[Cr_index];
        }
    }
    
    return ycbcr_image;
}


void init_constant_ycbcr_matrix() {
    hipMemcpyToSymbol(HIP_SYMBOL(device_YCbCr_matrix), YCbCr_matrix, sizeof(YCbCr_matrix));
    hipMemcpyToSymbol(HIP_SYMBOL(device_shift_vector), shift_vector, sizeof(shift_vector));
}

__global__ void RGB_2_YCbCr_kernel(int* rgb_image, float* ycbcr_image, int height, int width, int channels){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float device_YCbCr_matrix[3][3] = {{0.257, 0.504, 0.098},
                                         {-0.148, -0.291, 0.439},
                                         {0.439, -0.368, -0.071}};

    float device_shift_vector[3] = {16.0, 128.0, 128.0};

    int index = y * width * channels + x * channels;
    int R = rgb_image[index + 0];
    int G = rgb_image[index + 1];
    int B = rgb_image[index + 2];

    for (int c = 0; c < channels; c++)
    {
        ycbcr_image[index + c] = R * device_YCbCr_matrix[c][0] + 
                                 G * device_YCbCr_matrix[c][1] + 
                                 B * device_YCbCr_matrix[c][2] + 
                                 device_shift_vector[c];
    }
}

__global__ void chrominance_subsample_kernel(float* ycbcr_image, float* subsampled_image, int height, int width, int channels){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int CbCr_height = height / 2;
    int CbCr_width = width / 2;
    int origin_index = y * width * channels + x * channels;
    int Y_index = y * width + x;

    subsampled_image[Y_index] = ycbcr_image[origin_index];

    if(y % 2 == 0 && x % 2 == 0){
        int Cb_index = height * width + (y / 2) * CbCr_width + (x / 2);
        int Cr_index = height * width + CbCr_height * CbCr_width + (y / 2) * CbCr_width + (x / 2);
        subsampled_image[Cb_index] = ycbcr_image[origin_index + 1];
        subsampled_image[Cr_index] = ycbcr_image[origin_index + 2];
    }
}
