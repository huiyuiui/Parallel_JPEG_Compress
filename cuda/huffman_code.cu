#include "hip/hip_runtime.h"
#include "huffman_code.h"
#include <omp.h>

using namespace std;

vector<pair<int, int>> rle_compress(const int* data, int size) {
    vector<pair<int, int>> compressedData;

    int zeroCount = 0;

    for (int i = 0; i < size; ++i) {
        if (data[i] == 0) {
            zeroCount++;
        } else {
            compressedData.push_back({zeroCount, data[i]});
            zeroCount = 0;
        }
    }

   
    if (zeroCount > 0) {
        compressedData.push_back({-1, -1}); // EOB 標記
    }

    return compressedData;
}
int* rle_decompress(const vector<pair<int, int>> compressedData, int targetLength) {
    int* decompressedData = new int[targetLength];
    int currentIndex = 0;

    for (const auto& pair : compressedData) {
        if (pair.first == -1 && pair.second == -1) {
            while (currentIndex < targetLength) {
                decompressedData[currentIndex++] = 0;
            }
            break;
        } else {
            for (int i = 0; i < pair.first && currentIndex < targetLength; ++i) {
                decompressedData[currentIndex++] = 0;
            }
            decompressedData[currentIndex++] = pair.second;
        }
    }

    return decompressedData;
}


HuffmanNode* buildHuffmanTree(const unordered_map<int, int>& frequencies) {
    priority_queue<HuffmanNode*, vector<HuffmanNode*>, Compare> pq;

    for (const auto& pair : frequencies) {
        pq.push(new HuffmanNode(pair.first, pair.second));
    }

    while (pq.size() > 1) {
        HuffmanNode* left = pq.top();
        pq.pop();
        HuffmanNode* right = pq.top();
        pq.pop();

        
        HuffmanNode* merged = new HuffmanNode(-1, left->freq + right->freq);

        merged->left = (left->freq >= right->freq) ? right : left;
        merged->right = (left->freq >= right->freq) ? left : right;

        pq.push(merged);
    }
    HuffmanNode* root = pq.top();
    return root; 
}


void generateCodes(HuffmanNode* node, const string& prefix, unordered_map<int, string>& codebook) {
    if (!node) return;

    if (node->left == NULL && node->right == NULL) {
        codebook[node->value] = prefix;
    } 
    else {
        generateCodes(node->left, prefix + "0", codebook);
        generateCodes(node->right, prefix + "1", codebook);
    }
}


// unordered_map<int, int> calculateFrequencies(const int* data, int size) {
//     unordered_map<int, int> frequencies;

//     for (int i = 0; i < size; ++i) {
//         frequencies[data[i]]++;
//     }

//     return frequencies;
// }

unordered_map<int, int> calculateFrequencies(const int* data, int size) {
    int total_num = omp_get_max_threads();
    vector<unordered_map<int, int>> threadFrequencies(total_num);

    #pragma omp parallel
    {
        int threadId = omp_get_thread_num();
        for (int i = threadId; i < size; i += total_num) {
            threadFrequencies[threadId][data[i]]++;
        }
    }

    unordered_map<int, int> frequencies;
    for (const unordered_map<int, int>& threadFrequency : threadFrequencies) {
        for (const pair<int, int>& pair : threadFrequency) {
            frequencies[pair.first] += pair.second;
        }
    }

    return frequencies;
}

// string encodeData(const int* data, int size, const unordered_map<int, string>& codebook) {
//     string encodedData;
//     for (int i = 0; i < size; ++i) {
//         auto it = codebook.find(data[i]);
//         if (it != codebook.end()) {
//             encodedData += it->second;
//         } 
//         else {
//             cerr << "Error: Value " << data[i] << " not found in codebook." << endl;
//             exit(EXIT_FAILURE);
//         }
//     }
//     return encodedData;
// }
string encodeData(const int* data, int size, const unordered_map<int, string>& codebook) {
    int total_num = omp_get_max_threads();

    // Precompute chunk boundaries for each thread
    vector<int> chunk_starts(total_num), chunk_sizes(total_num);
    int base_size = size / total_num;
    int remainder = size % total_num;
    int offset = 0;

    for (int t = 0; t < total_num; t++) {
        int chunk_size = base_size + (t < remainder ? 1 : 0);
        chunk_starts[t] = offset;
        chunk_sizes[t] = chunk_size;
        offset += chunk_size;
    }

    vector<string> threadResults(total_num);

    #pragma omp parallel
    {
        int threadId = omp_get_thread_num();
        int start = chunk_starts[threadId];
        int end   = start + chunk_sizes[threadId];

        // If we had a direct indexing array for codebook, we could do something like:
        // const std::string* codeArray = ... // direct indexing from value
        // Then we'd avoid the map lookup.

        // Just parallel; no vectorization here since it's dictionary lookup
        string localEncoded;
        localEncoded.reserve((end - start) * 8); // guess capacity to reduce reallocations

        for (int i = start; i < end; i++) {
            auto it = codebook.find(data[i]);
            // If codebook lookup is O(1) average, this is fine. For direct indexing:
            // localEncoded += codeArray[data[i]];

            if (it != codebook.end()) {
                localEncoded += it->second;
            } else {
                // Handle error or missing code
            }
        }

        threadResults[threadId] = move(localEncoded);
    }

    // Combine results
    string encodedData;
    // Precompute size for efficiency
    size_t totalLength = 0;
    for (auto &res : threadResults) totalLength += res.size();
    encodedData.reserve(totalLength);

    for (auto &res : threadResults) {
        encodedData += move(res);
    }

    return encodedData;
}
pair<string, unordered_map<int, string>> huffman_encode(const int* data, int size) {

    auto compressedData = rle_compress(data, size);
    vector<int> rleFlattened;
    for (const auto& pair : compressedData) {
        rleFlattened.push_back(pair.first);
        rleFlattened.push_back(pair.second);
    }
    auto frequencies = calculateFrequencies(rleFlattened.data(), rleFlattened.size());
    
    HuffmanNode* root = buildHuffmanTree(frequencies);
    unordered_map<int, string> codebook;
    generateCodes(root, "", codebook);

    string encodedData = encodeData(rleFlattened.data(), rleFlattened.size(), codebook);

    return {encodedData, codebook};
}

int* huffman_decode(const string& encodedData, const unordered_map<int, string>& codebook, int outSize) {
    
    unordered_map<string, int> reverseCodebook;
    for (const auto& pair : codebook) {
        reverseCodebook[pair.second] = pair.first;
    }

    // Step 2: Huffman 解碼展平數據
    vector<int> rleFlattened;
    string currentCode;
    for (char bit : encodedData) {
        currentCode += bit;
        if (reverseCodebook.find(currentCode) != reverseCodebook.end()) {
            rleFlattened.push_back(reverseCodebook[currentCode]);
            currentCode.clear();
        }
    }

    vector<pair<int, int>> compressedData;
    for (int i = 0; i < static_cast<int>(rleFlattened.size()); i += 2) {
        compressedData.push_back({rleFlattened[i], rleFlattened[i + 1]});
    }

    return rle_decompress(compressedData, outSize);
}