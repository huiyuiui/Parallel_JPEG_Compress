#include "hip/hip_runtime.h"
#include "quantization.h"
#include <math.h>
#include <immintrin.h>
#include <iostream>
#include "utility.h"

using namespace std;

__constant__ int device_lumi_qtable[8][8];
__constant__ int device_chromi_qtable[8][8];
__constant__ float device_scale_factor;

int* quantization(float* dct_image, int height, int width){
    int CbCr_height = height / 2;
    int CbCr_width = width / 2;
    int* quantized_image = new int[height * width + 2 * CbCr_height * CbCr_width];

    for (int y = 0; y < height; y+=8)
    {
        for (int x = 0; x < width; x+=8)
        {
            for (int i = 0; i < 8; i++)
            {
                for (int j = 0; j < 8; j++)
                {
                    int Y_index = (y + i) * width + (x + j);
                    quantized_image[Y_index] = static_cast<int>(round(dct_image[Y_index] / Luminance_Qtable[i][j]));

                    if(y < CbCr_height && x < CbCr_width){
                        int Cb_index = height * width + (y + i) * CbCr_width + (x + j);
                        int Cr_index = height * width + CbCr_height * CbCr_width + (y + i) * CbCr_width + (x + j);
                        quantized_image[Cb_index] = static_cast<int>(round(dct_image[Cb_index] / Chrominance_Qtable[i][j]));
                        quantized_image[Cr_index] = static_cast<int>(round(dct_image[Cr_index] / Chrominance_Qtable[i][j]));
                    }
                }
            }
        }
    }

    return quantized_image;
}

int* dequantization(int* idct_image, int height, int width){
    int CbCr_height = height / 2;
    int CbCr_width = width / 2;
    int* dequantized_image = new int[height * width + 2 * CbCr_height * CbCr_width];

    for (int y = 0; y < height; y+=8)
    {
        for (int x = 0; x < width; x+=8)
        {
            for (int i = 0; i < 8; i++)
            {
                for (int j = 0; j < 8; j++)
                {
                    int Y_index = (y + i) * width + (x + j);
                    dequantized_image[Y_index] = idct_image[Y_index] * (Luminance_Qtable[i][j] * scale_factor);

                    if(y < CbCr_height && x < CbCr_width){
                        int Cb_index = height * width + (y + i) * CbCr_width + (x + j);
                        int Cr_index = height * width + CbCr_height * CbCr_width + (y + i) * CbCr_width + (x + j);
                        dequantized_image[Cb_index] = idct_image[Cb_index] * Chrominance_Qtable[i][j];
                        dequantized_image[Cr_index] = idct_image[Cr_index] * Chrominance_Qtable[i][j];
                    }
                }
            }
        }
    }

    return dequantized_image;
}


void init_constant_qtable(){
    hipMemcpyToSymbol(HIP_SYMBOL(device_lumi_qtable), Luminance_Qtable, sizeof(Luminance_Qtable));
    hipMemcpyToSymbol(HIP_SYMBOL(device_chromi_qtable), Chrominance_Qtable, sizeof(Chrominance_Qtable));
    hipMemcpyToSymbol(HIP_SYMBOL(device_scale_factor), &scale_factor, sizeof(scale_factor));
}

/* BlockSize(8, 8) version*/
// __global__ void quantization_kernel(float* dct_image, int* quantized_image, int height, int width){
//     int i = threadIdx.y;
//     int j = threadIdx.x;
//     int x = blockIdx.x * blockDim.x + j;
//     int y = blockIdx.y * blockDim.y + i;

//     int CbCr_height = height / 2;
//     int CbCr_width = width / 2;
//     int Y_index = y * width + x;
//     quantized_image[Y_index] = static_cast<int>(roundf(dct_image[Y_index] / device_lumi_qtable[i][j]));

//     if(y < CbCr_height && x < CbCr_width){
//         int Cb_index = height * width + y * CbCr_width + x;
//         int Cr_index = height * width + CbCr_height * CbCr_width + y * CbCr_width + x;
//         quantized_image[Cb_index] = static_cast<int>(roundf(dct_image[Cb_index] / device_chromi_qtable[i][j]));
//         quantized_image[Cr_index] = static_cast<int>(roundf(dct_image[Cr_index] / device_chromi_qtable[i][j]));
//     }
// }

/* BlockSize(32, 32) version*/
__global__ void quantization_kernel(float* dct_image, int* quantized_image, int height, int width){
    int i = threadIdx.y;
    int j = threadIdx.x;
    int x = blockIdx.x * blockDim.x + j;
    int y = blockIdx.y * blockDim.y + i;

    int CbCr_height = height / 2;
    int CbCr_width = width / 2;
    int Y_index = y * width + x;
    quantized_image[Y_index] = static_cast<int>(roundf(dct_image[Y_index] / (device_lumi_qtable[i % 8][j % 8] * device_scale_factor)));

    if(y < CbCr_height && x < CbCr_width){
        int Cb_index = height * width + y * CbCr_width + x;
        int Cr_index = height * width + CbCr_height * CbCr_width + y * CbCr_width + x;
        quantized_image[Cb_index] = static_cast<int>(roundf(dct_image[Cb_index] / device_chromi_qtable[i % 8][j % 8]));
        quantized_image[Cr_index] = static_cast<int>(roundf(dct_image[Cr_index] / device_chromi_qtable[i % 8][j % 8]));
    }
}