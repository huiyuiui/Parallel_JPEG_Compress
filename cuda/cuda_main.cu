#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <assert.h>
#include <time.h>
#include <iostream>
#include "utility.h"
#include "png_io.h"
#include "color_space.h"
#include "quantization.h"
#include "DCT.h"
#include "huffman_code.h"

using namespace std;

//======================
#define DEV_NO 0
hipDeviceProp_t prop;

int main(int argc, char* argv[]) {
    hipGetDeviceProperties(&prop, DEV_NO);
    printf("maxThreasPerBlock = %d, sharedMemPerBlock = %d\n", prop.maxThreadsPerBlock, prop.sharedMemPerBlock);
    
    assert(argc == 2);
    const string filename = argv[1];

    // read image
    Image img = read_png(filename);

    // init variable
    int height = img.height;
    int width = img.width;
    int channels = img.channels;
    int full_size = height * width * channels;
    int half_size = height * width + 2 * (height / 2 * width / 2);
    struct timespec start, end;
    double elapsed_time;

    // declare array
    int* host_full_img_i, *host_half_img_i;
    float* host_full_img_f, *host_half_img_f;
    int* dev_full_img_i, *dev_half_img_i;
    float* dev_full_img_f, *dev_half_img_f;
    
    // allocate memory
    host_full_img_i = Image_2_pointer(img);
    host_full_img_f = new float[full_size];
    host_half_img_i = new int[half_size];
    host_half_img_f = new float[half_size];
    hipMalloc((void**)&dev_full_img_i, full_size * sizeof(int));
    hipMalloc((void**)&dev_full_img_f, full_size * sizeof(float));
    hipMalloc((void**)&dev_half_img_i, half_size * sizeof(int));
    hipMalloc((void**)&dev_half_img_f, half_size * sizeof(float));

    // memory copy from host to device
    hipMemcpy(dev_full_img_i, host_full_img_i, full_size * sizeof(int), hipMemcpyHostToDevice);
    init_constant_ycbcr_matrix();
    init_constant_qtable();

    // kernel parameters
    int BLOCK_SIZE = 32;
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blockNum((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    /* Compression */
    clock_gettime(CLOCK_MONOTONIC, &start);
    // step 1: convert RGB to YCbCr
    RGB_2_YCbCr_kernel<<<blockNum, blockSize>>>(dev_full_img_i, dev_full_img_f, height, width, channels);

    // step 2: chrominance subsample
    chrominance_subsample_kernel<<<blockNum, blockSize>>>(dev_full_img_f, dev_half_img_f, height, width, channels);
    
    // step 3: DCT
    // TODO:
    hipMemcpy(host_half_img_f, dev_half_img_f, half_size * sizeof(float), hipMemcpyDeviceToHost);
    // float* dct_image = DCT(host_half_img_f, height, width);
    float* dct_image = DCT_cuda(host_half_img_f, height, width);
    hipMemcpy(dev_half_img_f, dct_image, half_size * sizeof(float), hipMemcpyHostToDevice);

    // step 4: quantization
    // dim3 quantizedBlockSize(8, 8);
    // dim3 quantizedBlockNum((width + 8 - 1) / 8, (height + 8 - 1) / 8);
    quantization_kernel<<<blockNum, blockSize>>>(dev_half_img_f, dev_half_img_i, height, width);

    clock_gettime(CLOCK_MONOTONIC, &end);
    elapsed_time = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
    printf("Elapsed time: %f seconds\n", elapsed_time);

    // memory copy from device back to host
    hipMemcpy(host_half_img_i, dev_half_img_i, half_size * sizeof(int), hipMemcpyDeviceToHost);
    
    // step 5: huffman code
    auto [encoded_image, codebook] = huffman_encode(host_half_img_i, height * width + 2 * height / 2 * width / 2);

    /* Decompression */
    // step 1: huffman decoding
    int *decoded_image = huffman_decode(encoded_image, codebook , height * width + 2 * height / 2 * width / 2);

    // step 2: dequantization
    int* dequantized_image = dequantization(host_half_img_i, height, width);

    // step 3: IDCT
    float* idct_image = iDCT(dequantized_image, height, width);

    // step 4: chrominance upsample
    float* ycbcr_image = chrominance_upsample(idct_image, height, width, channels);

    // YCbCr to image
    float *rgb_image = YcbCr_2_RGB(ycbcr_image, height, width, channels);

    float psnr = PSNR(img, rgb_image);
    float subsample_compressed_ratio = compression_ratio(full_size, height * width + 2 * height / 2 * width / 2);
    float huffman_compressed_ratio = compression_ratio(full_size * sizeof(int) * 8, encoded_image.length());

    cout << "Compressed PSNR: " << psnr << endl;
    cout << "Compressed ratio after subsample: " << subsample_compressed_ratio << endl;
    cout << "Compressed ratio after huffman encode: " << huffman_compressed_ratio << endl;

    // recover image
    Image rgb_img = pointer_2_Image(rgb_image, height, width, channels);

    write_png("../assets/output.png", rgb_img);

    return 0;
}