#include <iostream>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
using namespace std;

__constant__ float device_temp;

__global__ void DCT_cuda_cal(int N, float *input, float *output, int stride) {
    int offset = blockIdx.x * N * stride + blockIdx.y * N;
    int u = threadIdx.x, v = threadIdx.y;

    __shared__ float cos_values[8][8];
    __shared__ float S_output[8][8];
    __shared__ float S_input[8][8];

    cos_values[u][v] = cos((2 * u + 1) * v * M_PI / (2 * N));
    S_input[u][v] = input[offset + u * stride + v];
    S_output[u][v] = 0;
    __syncthreads();

    for (int x = 0; x < N; x++) {
        for (int y = 0; y < N; y++) {
            S_output[u][v] += S_input[x][y] * cos_values[x][u] * cos_values[y][v];
        }
    }
    S_output[u][v] *= 2.0 / N;

    if (u == 0) {
        S_output[0][v] *= device_temp;
    }
    if (v == 0) {
        S_output[u][0] *= device_temp;
    }

    output[offset + u * stride + v] = S_output[u][v];
}

void DCT_cuda(float *input, float *output, int height, int width) {
    const int N = 8;
    const int CbCr_height = height / 2, CbCr_width = width / 2;

    float temp = 1 / sqrt(2.0);
    hipMemcpyToSymbol(HIP_SYMBOL(device_temp), &temp, sizeof(float));

    dim3 num_threads(N, N);
    dim3 num_blocks(height / N, width / N);
    dim3 num_blocks_CbCr(CbCr_height / N, CbCr_width / N);

    DCT_cuda_cal<<<num_blocks, num_threads>>>(N, input, output, width);
    DCT_cuda_cal<<<num_blocks_CbCr, num_threads>>>(N, input + height * width, output + height * width, CbCr_width);
    DCT_cuda_cal<<<num_blocks_CbCr, num_threads>>>(N, input + height * width + CbCr_height * CbCr_width, output + height * width + CbCr_height * CbCr_width, CbCr_width);

    return;
}

void DCT_cal(int N, float *input, float *output, int stride) {
    float cos_values[N][N];
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cos_values[i][j] = cos((2 * i + 1) * j * M_PI / (2 * N));
        }
    }

    for (int u = 0; u < N; u++) {
        for (int v = 0; v < N; v++) {
            output[u * stride + v] = 0;
            for (int x = 0; x < N; x++) {
                for (int y = 0; y < N; y++) {
                    output[u * stride + v] += input[x * stride + y] * cos_values[x][u] * cos_values[y][v];
                }
            }
            output[u * stride + v] *= 2.0 / N;
        }
    }

    float temp = 1 / sqrt(2.0);
    for (int i = 0; i < N; i++) {
        output[i * stride] *= temp;
        output[i] *= temp;
    }
}

float* DCT(float *input, int height, int width) {
    const int N = 8;
    const int CbCr_height = height / 2, CbCr_width = width / 2;

    float *output = new float[height * width + 2 * CbCr_height * CbCr_width];
    // for Y
    for (int i = 0; i < height/N; i++) {
        for (int j = 0; j < width/N; j++) {
            DCT_cal(N, input+i*N*width+j*N, output+i*N*width+j*N, width);
        }
    }
    // for Cb and Cr
    for (int i = 0; i < CbCr_height/N; i++) {
        for (int j = 0; j < CbCr_width/N; j++) {
            DCT_cal(N, input+height*width+i*N*CbCr_width+j*N, output+height*width+i*N*CbCr_width+j*N, CbCr_width);
            DCT_cal(N, input+height*width+CbCr_height*CbCr_width+i*N*CbCr_width+j*N, output+height*width+CbCr_height*CbCr_width+i*N*CbCr_width+j*N, CbCr_width);
        }
    }

    return output;
}

void iDCT_cal(int N, int *input, float *output, int stride) {
    float alpha_u, alpha_v, temp = 1 / sqrt(2.0);
    float cos_values[N][N];
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cos_values[i][j] = cos((2 * i + 1) * j * M_PI / (2 * N));
        }
    }

    for (int x = 0; x < N; x++) {
        for (int y = 0; y < N; y++) {
            output[x * stride + y] = 0;
            for (int u = 0; u < N; u++) {
                alpha_u = (u == 0) ? temp : 1;
                for (int v = 0; v < N; v++) {
                    alpha_v = (v == 0) ? temp : 1;
                    output[x * stride + y] += alpha_u * alpha_v * input[u * stride + v] * cos_values[x][u] * cos_values[y][v];
                }
            }
            output[x * stride + y] *= 2.0 / N;
        }
    }
}

float* iDCT(int *input, int height, int width) {
    const int N = 8;
    const int CbCr_height = height / 2, CbCr_width = width / 2;

    float *output = new float[height * width + 2 * CbCr_height * CbCr_width];
    // for Y
    for (int i = 0; i < height/N; i++) {
        for (int j = 0; j < width/N; j++) {
            iDCT_cal(N, input+i*N*width+j*N, output+i*N*width+j*N, width);
        }
    }
    // for Cb and Cr
    for (int i = 0; i < CbCr_height/N; i++) {
        for (int j = 0; j < CbCr_width/N; j++) {
            iDCT_cal(N, input+height*width+i*N*CbCr_width+j*N, output+height*width+i*N*CbCr_width+j*N, CbCr_width);
            iDCT_cal(N, input+height*width+CbCr_height*CbCr_width+i*N*CbCr_width+j*N, output+height*width+CbCr_height*CbCr_width+i*N*CbCr_width+j*N, CbCr_width);
        }
    }

    return output;
}