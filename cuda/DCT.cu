#include <iostream>
#include <cmath>
#include <chrono>
#include <omp.h>
#include <cstdlib>
#include <immintrin.h>
#include <hip/hip_runtime.h>
using namespace std;

__constant__ float device_temp;

__global__ void DCT_cuda_cal(int N, float *input, float *output, int stride) {
    int offset = blockIdx.x * N * stride + blockIdx.y * N;
    int u = threadIdx.x, v = threadIdx.y;

    __shared__ float cos_values[8][8];
    cos_values[u][v] = cos((2 * u + 1) * v * M_PI / (2 * N));
    __syncthreads();

    output[offset + u * stride + v] = 0;
    for (int x = 0; x < N; x++) {
        for (int y = 0; y < N; y++) {
            output[offset + u * stride + v] += input[offset + x * stride + y] * cos_values[x][u] * cos_values[y][v];
        }
    }
    output[offset + u * stride + v] *= 2.0 / N;

    if (u == 0) {
        output[offset + v] *= device_temp;
    }
    if (v == 0) {
        output[offset + u * stride] *= device_temp;
    }
}

float* DCT_cuda(float *input, int height, int width) {
    const int N = 8;
    const int CbCr_height = height / 2, CbCr_width = width / 2;

    float *output = new float[height * width + 2 * CbCr_height * CbCr_width];

    float *device_input, *device_output, temp = 1 / sqrt(2.0);
    hipMalloc(&device_input, (height * width + 2 * CbCr_height * CbCr_width) * sizeof(float));
    hipMalloc(&device_output, (height * width + 2 * CbCr_height * CbCr_width) * sizeof(float));
    hipMemcpy(device_input, input, (height * width + 2 * CbCr_height * CbCr_width) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(device_temp), &temp, sizeof(float));

    dim3 num_threads(N, N);
    dim3 num_blocks(height / N, width / N);
    dim3 num_blocks_CbCr(CbCr_height / N, CbCr_width / N);

    DCT_cuda_cal<<<num_blocks, num_threads>>>(N, device_input, device_output, width);
    DCT_cuda_cal<<<num_blocks_CbCr, num_threads>>>(N, device_input + height * width, device_output + height * width, CbCr_width);
    DCT_cuda_cal<<<num_blocks_CbCr, num_threads>>>(N, device_input + height * width + CbCr_height * CbCr_width, device_output + height * width + CbCr_height * CbCr_width, CbCr_width);

    hipMemcpy(output, device_output, (height * width + 2 * CbCr_height * CbCr_width) * sizeof(float), hipMemcpyDeviceToHost);

    return output;
}

void DCT_cal(int N, float *input, float *output, int stride) {
    float cos_values[N][N];
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cos_values[i][j] = cos((2 * i + 1) * j * M_PI / (2 * N));
        }
    }

    for (int u = 0; u < N; u++) {
        for (int v = 0; v < N; v++) {
            output[u * stride + v] = 0;
            for (int x = 0; x < N; x++) {
                for (int y = 0; y < N; y++) {
                    output[u * stride + v] += input[x * stride + y] * cos_values[x][u] * cos_values[y][v];
                }
            }
            output[u * stride + v] *= 2.0 / N;
        }
    }

    float temp = 1 / sqrt(2.0);
    for (int i = 0; i < N; i++) {
        output[i * stride] *= temp;
        output[i] *= temp;
    }
}

float* DCT(float *input, int height, int width) {
    const int N = 8;
    const int CbCr_height = height / 2, CbCr_width = width / 2;

    float *output = new float[height * width + 2 * CbCr_height * CbCr_width];
    // for Y
    for (int i = 0; i < height/N; i++) {
        for (int j = 0; j < width/N; j++) {
            DCT_cal(N, input+i*N*width+j*N, output+i*N*width+j*N, width);
        }
    }
    // for Cb and Cr
    for (int i = 0; i < CbCr_height/N; i++) {
        for (int j = 0; j < CbCr_width/N; j++) {
            DCT_cal(N, input+height*width+i*N*CbCr_width+j*N, output+height*width+i*N*CbCr_width+j*N, CbCr_width);
            DCT_cal(N, input+height*width+CbCr_height*CbCr_width+i*N*CbCr_width+j*N, output+height*width+CbCr_height*CbCr_width+i*N*CbCr_width+j*N, CbCr_width);
        }
    }

    return output;
}

void iDCT_cal(int N, int *input, float *output, int stride) {
    float alpha_u, alpha_v, temp = 1 / sqrt(2.0);
    float cos_values[N][N];
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            cos_values[i][j] = cos((2 * i + 1) * j * M_PI / (2 * N));
        }
    }

    for (int x = 0; x < N; x++) {
        for (int y = 0; y < N; y++) {
            output[x * stride + y] = 0;
            for (int u = 0; u < N; u++) {
                alpha_u = (u == 0) ? temp : 1;
                for (int v = 0; v < N; v++) {
                    alpha_v = (v == 0) ? temp : 1;
                    output[x * stride + y] += alpha_u * alpha_v * input[u * stride + v] * cos_values[x][u] * cos_values[y][v];
                }
            }
            output[x * stride + y] *= 2.0 / N;
        }
    }
}

float* iDCT(int *input, int height, int width) {
    const int N = 8;
    const int CbCr_height = height / 2, CbCr_width = width / 2;

    float *output = new float[height * width + 2 * CbCr_height * CbCr_width];
    // for Y
    for (int i = 0; i < height/N; i++) {
        for (int j = 0; j < width/N; j++) {
            iDCT_cal(N, input+i*N*width+j*N, output+i*N*width+j*N, width);
        }
    }
    // for Cb and Cr
    for (int i = 0; i < CbCr_height/N; i++) {
        for (int j = 0; j < CbCr_width/N; j++) {
            iDCT_cal(N, input+height*width+i*N*CbCr_width+j*N, output+height*width+i*N*CbCr_width+j*N, CbCr_width);
            iDCT_cal(N, input+height*width+CbCr_height*CbCr_width+i*N*CbCr_width+j*N, output+height*width+CbCr_height*CbCr_width+i*N*CbCr_width+j*N, CbCr_width);
        }
    }

    return output;
}